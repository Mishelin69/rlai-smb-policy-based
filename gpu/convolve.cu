
#include <hip/hip_runtime.h>

#include <stdio.h>

#ifndef MAX_KERNEL_SIZE
#define MAX_KERNEL_SIZE 12*12
#endif

__global__
void convolve_v1(const float* k, const float* m, float* o, int kx, int mx, int ox) {

    //using __shared__ because I saw it somewhere and made sense at the time idk if it'll actually be beneficial
    //but local cache is 48k so most likely it won't be a problem but still better than accessing global all the time
    __shared__ float sharedKernel[MAX_KERNEL_SIZE];
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //load the kernel and check if x and y are in bound of kernel
    //then wait for this to be loaded and in sync 
    //to make sure that the kernel is properly loaded in
    if (y*kx + x < kx*kx) {
        sharedKernel[y*kx + x] = k[y*kx + x];
    }
    __syncthreads();

    //we only expect square matrices
    if (x < ox && y < ox) {

        float sum = 0;

        for (size_t rows = 0; rows < kx; ++rows) {
            for (size_t cols = 0; cols < kx; ++cols) {
                sum += sharedKernel[kx*rows + cols] * m[y*mx + rows*mx + cols + x];
            }
        }

        o[y*ox + x] += sum;
    }
}

__global__
void convolve_v1_ReLU(const float* k, const float* m, float* o, int kx, int mx, int ox) {

    //using __shared__ because I saw it somewhere and made sense at the time idk if it'll actually be beneficial
    //but local cache is 48k so most likely it won't be a problem but still better than accessing global all the time
    __shared__ float sharedKernel[MAX_KERNEL_SIZE];
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //load the kernel and check if x and y are in bound of kernel
    //then wait for this to be loaded and in sync 
    //to make sure that the kernel is properly loaded in
    if (y*kx + x < kx*kx) {
        sharedKernel[y*kx + x] = k[y*kx + x];
    }
    __syncthreads();

    //we only expect square matrices
    if (x < ox && y < ox) {

        float sum = 0;

        for (size_t rows = 0; rows < kx; ++rows) {
            for (size_t cols = 0; cols < kx; ++cols) {
                sum += sharedKernel[kx*rows + cols] * m[y*mx + rows*mx + cols + x];
            }
        }

        o[y*ox + x] += (sum >= 0) ? sum : 0;
    }
}

__global__
void convolve_v1_Sigmoid(const float* k, const float* m, float* o, int kx, int mx, int ox) {

    //using __shared__ because I saw it somewhere and made sense at the time idk if it'll actually be beneficial
    //but local cache is 48k so most likely it won't be a problem but still better than accessing global all the time
    __shared__ float sharedKernel[MAX_KERNEL_SIZE];
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //load the kernel and check if x and y are in bound of kernel
    //then wait for this to be loaded and in sync 
    //to make sure that the kernel is properly loaded in
    if (y*kx + x < kx*kx) {
        sharedKernel[y*kx + x] = k[y*kx + x];
    }
    __syncthreads();

    //we only expect square matrices
    if (x < ox && y < ox) {

        float sum = 0;

        for (size_t rows = 0; rows < kx; ++rows) {
            for (size_t cols = 0; cols < kx; ++cols) {
                sum += sharedKernel[kx*rows + cols] * m[y*mx + rows*mx + cols + x];
            }
        }

        o[y*ox + x] += 1.0 / (1.0 + exp(-sum));
    }
}

__global__
void batched_convolve_v1_ReLU(const float* k, const float* m, float* o, 
        int kx, int mx, int ox, const size_t b_size, const size_t n_elms) {

    //using __shared__ because I saw it somewhere and made sense at the time idk if it'll actually be beneficial
    //but local cache is 48k (actually around 100k on my GPU but the point still stands) 
    //so most likely it won't be a problem but still better than accessing global all the time
    extern __shared__ float sharedKernel[];
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //load the kernel and check if x and y are in bound of kernel
    //then wait for this to be loaded and in sync 
    //to make sure that the kernel is properly loaded in
    if (x < kx && y < kx*n_elms) {
        sharedKernel[y*kx + x] = k[y*kx + x];
    }
    __syncthreads();

    //we only expect square matrices
    if (x < ox && y < ox * n_elms) {

        //Index of the element inside batch
        const size_t nth_elm = (y*ox + x) / b_size;
        float sum = 0;

        for (size_t rows = 0; rows < kx; ++rows) {
            for (size_t cols = 0; cols < kx; ++cols) {

                const size_t kernel_index = nth_elm*kx*kx + kx*rows + cols;
                const size_t mat_index = y*mx + rows*mx + cols + x;

                sum += sharedKernel[kernel_index] * m[mat_index];
            }
        }

        o[y*ox + x] += (sum >= 0) ? sum : 0;
    }
}

__global__
void batched_convolve_v1_Sigmoid(const float* k, const float* m, float* o, 
        int kx, int mx, int ox, const size_t b_size, const size_t n_elms) {

    //using __shared__ because I saw it somewhere and made sense at the time idk if it'll actually be beneficial
    //but local cache is 48k (actually around 100k on my GPU but the point still stands) 
    //so most likely it won't be a problem but still better than accessing global all the time
    extern __shared__ float sharedKernel[];
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //load the kernel and check if x and y are in bound of kernel
    //then wait for this to be loaded and in sync 
    //to make sure that the kernel is properly loaded in
    if (x < kx && y < kx*n_elms) {
        sharedKernel[y*kx + x] = k[y*kx + x];
    }
    __syncthreads();

    //we only expect square matrices
    if (x < ox && y < ox * n_elms) {

        //Index of the element inside batch
        const size_t nth_elm = (y*ox + x) / b_size;
        float sum = 0;

        for (size_t rows = 0; rows < kx; ++rows) {
            for (size_t cols = 0; cols < kx; ++cols) {

                const size_t kernel_index = nth_elm*kx*kx + kx*rows + cols;
                const size_t mat_index = y*mx + rows*mx + cols + x;

                sum += sharedKernel[kernel_index] * m[mat_index];
            }

        }        

        o[y*ox + x] += 1.0 / (1.0 + exp(-sum));
    }
}

//supports x*y bacthes not only x*x ! :) pain
__global__
void batched_convolve_v2_ReLU(const float* k, const float* m, float* o, 
        int kx, int mx, int ox, const size_t b_size, const size_t n_elms, const size_t inputs) {

    //using __shared__ because I saw it somewhere and made sense at the time idk if it'll actually be beneficial
    //but local cache is 48k (actually around 100k on my GPU but the point still stands) 
    //so most likely it won't be a problem but still better than accessing global all the time
    extern __shared__ float sharedKernel[];
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //load the kernel and check if x and y are in bound of kernel
    //then wait for this to be loaded and in sync 
    //to make sure that the kernel is properly loaded in
    if (x < kx && y < kx*n_elms) {
        printf("KERNEL HIT \n");
        sharedKernel[y*kx + x] = k[y*kx + x];
    }
    __syncthreads();

    //we only expect square matrices
    if (x < ox && y < ox * n_elms) {

        //Index of the element inside batch
        const size_t nth_elm = (y*ox + x) / b_size;
        float sum = 0;

        for (size_t rows = 0; rows < kx; ++rows) {
            for (size_t cols = 0; cols < kx; ++cols) {
                for (size_t i = 0; i < inputs; ++i) {

                    const size_t kernel_index = nth_elm*kx*kx + kx*rows + cols;
                    const size_t mat_index = i*b_size + y*mx + rows*mx + cols + x;

                    sum += sharedKernel[kernel_index] * m[mat_index];
                }
            }
        }

        o[y*ox + x] += (sum >= 0) ? sum : 0;
    }
}

//supports x*y bacthes not only x*x ! :) pain
__global__
void batched_convolve_v2_Sigmoid(const float* k, const float* m, float* o, 
        int kx, int mx, int ox, const size_t b_size, const size_t n_elms, const size_t inputs) {

    //using __shared__ because I saw it somewhere and made sense at the time idk if it'll actually be beneficial
    //but local cache is 48k (actually around 100k on my GPU but the point still stands) 
    //so most likely it won't be a problem but still better than accessing global all the time
    extern __shared__ float sharedKernel[];
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    //load the kernel and check if x and y are in bound of kernel
    //then wait for this to be loaded and in sync 
    //to make sure that the kernel is properly loaded in
    if (x < kx && y < kx*n_elms) {
        sharedKernel[y*kx + x] = k[y*kx + x];
    }
    __syncthreads();

    //we only expect square matrices
    if (x < ox && y < ox * n_elms) {

        //Index of the element inside batch
        const size_t nth_elm = (y*ox + x) / b_size;
        float sum = 0;

        for (size_t rows = 0; rows < kx; ++rows) {
            for (size_t cols = 0; cols < kx; ++cols) {
                for (size_t i = 0; i < inputs; ++i) {

                    const size_t kernel_index = nth_elm*kx*kx + kx*rows + cols;
                    const size_t mat_index = i*b_size + y*mx + rows*mx + cols + x;

                    sum += sharedKernel[kernel_index] * m[mat_index];
                }
            }
        }

        o[y*ox + x] += 1.0 / (1.0 + exp(-sum));
    }
}
