#include "./dense_layer.hpp"
#include <hip/hip_runtime_api.h>
#include <iostream>

DenseLayer::DenseLayer(GPU::Device& gpu, float* cuda_w, float* cuda_b, const size_t neurons, 
        const size_t input, const GPU::ActivationFunction actv_func, const GPU::ActivationFunction der_actv_func)
    : gpu(gpu), input_shape(input), neurons(neurons), actv_func(actv_func), der_actv_func(der_actv_func) {

        this->mat_y = neurons;
        this->mat_x = input;
        this->biases = neurons;

        float* cudaMat = cuda_w; 

        int res = gpu.random_numbers(cudaMat, mat_y * mat_x);

        //!!res would be crazy but correct :D since only 0 evals as false (talking numbers ofc)
        //(negative numbers eval to true since they hold some value :| )
        if (res != 0) {
            std::cerr << "DenseLayer::DenseLayer() | Error: Error in initializing neurons!!" << std::endl; 
        }

        float* cudaBias = cuda_b;
        res = gpu.random_numbers(cudaBias, biases);

        if (res != 0) {
            std::cerr << "DenseLayer::DenseLayer() | Error: Error in initializing biases!!" << std::endl; 
        }
    }

void DenseLayer::init_self(GPU::Device& gpu, float* cuda_w, float* cuda_b, const size_t neurons, 
        const size_t input, const GPU::ActivationFunction actv_func, 
        const GPU::ActivationFunction der_actv_func) {

    this->gpu = gpu;
    this->input_shape = input;
    this->neurons = neurons;
    this->actv_func = actv_func;
    this->der_actv_func = der_actv_func;

    this->mat_y = neurons;
    this->mat_x = input;
    this->biases = neurons;

    float* cudaMat = cuda_w; 

    int res = gpu.random_numbers(cudaMat, mat_y * mat_x);

    //!!res would be crazy but correct :D since only 0 evals as false (talking numbers ofc)
    //(negative numbers eval to true since they hold some value :| )
    if (res != 0) {
        std::cerr << "DenseLayer::DenseLayer() | Error: Error in initializing neurons!!" << std::endl; 
    }

    float* cudaBias = cuda_b;
    res = gpu.random_numbers(cudaBias, biases);

    if (res != 0) {
        std::cerr << "DenseLayer::DenseLayer() | Error: Error in initializing biases!!" << std::endl; 
    }


}


void DenseLayer::passthrough(float* a, float* out, const hipStream_t stream) const noexcept {

    std::pair<size_t, size_t> out_shape = GPU::Device::calculate_new_mat_dims(mat_x, mat_y, input_shape, input_shape); 

    size_t out_y = out_shape.first;
    size_t out_x = out_shape.second;

    gpu.matmul_ver1_gpu(
            this->cudaMat,
            a,
            out,
            this->mat_y,
            this->mat_x,
            1,
            this->input_shape,
            out_y,
            out_x,
            this->actv_func,
            this->stream
            );

    gpu.matadd_ver1(
            this->cudaBias,
            a,
            out,
            this->biases,
            1,
            biases,
            1,
            out_y, //this should match but worst scenario I get an error :chomik_xmas:
            1,
            this->stream
            );

    hipStreamSynchronize(stream);
}

//fix this to do the correct thing :(
//yeah its not doing that buddy :)
//Ill need to adjust it, to be more modular or something else idk make either a diff 
//function or just do it differentlya (pain) Ill figure it out later (soon pain again)
void DenseLayer::gradient_calculation(const GPU::Tensor activations, 
        const GPU::Tensor gradient, GPU::Tensor out, const hipStream_t stream) const noexcept {

    gpu.matmul_ver1_gpu(
            gradient.dat_pointer, 
            cudaMat, 
            out.dat_pointer, 
            gradient.dat_x, 
            1, 
            1, 
            mat_y, 
            out.dat_x, 
            out.dat_y, 
            this->actv_func, 
            stream
            );

    //the end thing where you multiply by funciton derivartive the original output !:)
    gpu.matmul_elementwise(out, activations, out, stream, der_actv_func);

    hipStreamSynchronize(stream);
}
