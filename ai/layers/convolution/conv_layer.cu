#include "./conv_layer.hpp"
#include <hip/hip_runtime_api.h>
#include <iostream>

std::pair<uint32_t, uint32_t> ConvolutionalLayer::calc_output_size(
        const uint32_t kernel_x, const uint32_t kernel_y,
        const uint32_t input_x, const uint32_t input_y, const uint32_t kernel_shift) noexcept {

    const uint32_t _shft = kernel_shift - 1;

    const uint32_t out_x = input_x + 1 - (kernel_x + _shft);
    const uint32_t out_y = input_x + 1 - (kernel_x + _shft);

    return std::pair<uint32_t, uint32_t> { out_x, out_y };
}

ConvolutionalLayer::ConvolutionalLayer() {

}

ConvolutionalLayer::ConvolutionalLayer(GPU::Device& gpu, GPU::ActivationFunction actv_func,
            const uint32_t feature_maps, 
            const uint32_t input_chanels, const uint32_t kernel_dim, 
            const uint32_t input, float* cuda_w, float* cuda_b)
    : gpu(gpu), feature_maps(feature_maps), actv_func(actv_func),
    kernel_x(kernel_dim), kernel_y(kernel_dim), input_chanels(input_chanels) {

        //calculate the amount of memory needed (in bytes) to keep the stuff in
        const uint32_t kernel_size_bytes = sizeof(float) * kernel_dim * kernel_dim;
        const uint32_t mem_size = kernel_dim * kernel_dim * feature_maps * input_chanels;
        this->cuda_kernel_size = kernel_size_bytes;

        this->cuda_kernel = cuda_w;

        auto out = calc_output_size(kernel_dim, kernel_dim, input, input, 1);
        auto out_x = out.first;

        this->cuda_bias = cuda_b;

        for (uint32_t i = 0; i < feature_maps; ++i) {
            this->filters.push_back(ConvFilter {
                    this->cuda_kernel + input_chanels * kernel_x * kernel_y,
                    input_chanels
                    });
        }

        //this will be slow but whatever
        int res = gpu.random_numbers(this->cuda_kernel, input_chanels * feature_maps * kernel_x * kernel_y);

        if (res != 0) {
            std::cerr << "ConvolutionalLayer::ConvolutionalLayer() | Error while trying to initialize kernel data!" << std::endl;
            exit(-1);
        }

        res = gpu.random_numbers(this->cuda_bias, feature_maps * out_x * out_x);

        if (res != 0) {
            std::cerr << "ConvolutionalLayer::ConvolutionalLayer() | Error while trying to initialize bias data!" << std::endl;
            exit(-1);
        }
    }

void ConvolutionalLayer::deep_copy(const ConvolutionalLayer& original) {

    hipMemcpy(this->cuda_kernel, original.cuda_kernel, 
            sizeof(float) * input_chanels * feature_maps * kernel_x * kernel_y, hipMemcpyDeviceToDevice);

    hipMemcpy(this->cuda_bias, original.cuda_bias, 
            sizeof(float) * feature_maps, hipMemcpyDeviceToDevice);
}

//Ughhh God Im so lazy please finish this while Im away :) <3
//He did not finish it, I sadly had to do it myself :( not cool
void ConvolutionalLayer::init_self(GPU::Device& gpu, GPU::ActivationFunction func,
        const uint32_t feature_maps, 
        const uint32_t input_chanels, const uint32_t kernel_dim, 
        const uint32_t input, float* cuda_w, float* cuda_b) {

    this->feature_maps = feature_maps;
    this->input_chanels = input_chanels;
    this->gpu = gpu;
    this->actv_func = func;
    this->kernel_x = kernel_dim;
    this->kernel_y = kernel_dim;

    //calculate the amount of memory needed (in bytes) to keep the stuff in
    const uint32_t kernel_size_bytes = sizeof(float) * kernel_dim * kernel_dim;
    const uint32_t mem_size = kernel_dim * kernel_dim * feature_maps * input_chanels;
    this->cuda_kernel_size = kernel_size_bytes;

    this->cuda_kernel = cuda_w;

    auto out = calc_output_size(kernel_dim, kernel_dim, input, input, 1);
    auto out_x = out.first;

    this->cuda_bias = cuda_b;

    for (uint32_t i = 0; i < feature_maps; ++i) {
        this->filters.push_back(ConvFilter {
                this->cuda_kernel + input_chanels * kernel_x * kernel_y,
                input_chanels
                });
    }

    //this will be slow but whatever
    int res = gpu.random_numbers(this->cuda_kernel, input_chanels * feature_maps * kernel_x * kernel_y);

    if (res != 0) {
        std::cerr << "ConvolutionalLayer::ConvolutionalLayer() | Error while trying to initialize kernel data!" << std::endl;
        exit(-1);
    }

    res = gpu.random_numbers(this->cuda_bias, feature_maps);

    if (res != 0) {
        std::cerr << "ConvolutionalLayer::ConvolutionalLayer() | Error while trying to initialize bias data!" << std::endl;
        exit(-1);
    }
}



//fix
//now correctly does the convolution plus adds the bias :)
//at least it should do this, wherever this will be done correctly idk
//parameters:
//a: input
//b: the kernel
//out: the out pointer
//My little note some time layer, why b if b is member ????
//Im literraly creating new Tensor each loop so why b in the first place
//I guess Ill keep it in because I dont want it to break yk
//I honestly have no clue if it'll actually break but Im not 
//playing the devil over here JUST IN CASE! :-)
void ConvolutionalLayer::convolve(GPU::Tensor a, GPU::Tensor b, float* out, hipStream_t stream) const noexcept {

    const std::pair<uint32_t, uint32_t> out_dims = ConvolutionalLayer::calc_output_size(
            b.dat_x, b.dat_y, a.dat_x, a.dat_y, 1
            );

    const uint32_t dim_x = out_dims.first;
    const uint32_t dim_y = out_dims.second;

    //queue up jobs and wait for them to finish
    for (size_t i = 0; i < this->feature_maps; ++i) {

        this->gpu.conv_add(
                GPU::Tensor { 
                this->cuda_bias, 
                dim_x,
                1,//1 because same bias for everything 
                  //also bias is just 1D vector where its shape is (n,) 
                  //where n => number of output feature maps
                1 
                },

                GPU::Tensor {
                out + i*dim_x*dim_y,
                dim_x,
                dim_y,
                1
                }, i, stream);


        this->gpu.conv_ver2(
                a, 
                GPU::Tensor { 
                this->filters[i].cuda_kernels, 
                this->kernel_x,
                this->kernel_y, 
                this->filters[i].depth 
                },

                GPU::Tensor {
                out + i*dim_x*dim_y,
                dim_x,
                dim_y,
                1
                }, 0, stream);

    }

    //wait for the GPU to finish it's job (stream)
    //keep the data on the GPU tho
    //hipStreamSynchronize(stream);
}
